#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <chrono>
#include "image.h"
#include "misc.h"
#include "pnmfile.h"
#include "segment-image.cuh"
#include <hip/hip_runtime.h>

int main(int argc, char **argv) {
    // Verifica que se pasen los argumentos necesarios
    if (argc != 6) {
        fprintf(stderr, "usage: %s sigma k min input(ppm) output(ppm)\n", argv[0]);
        return 1;
    }

    // Lee los parámetros de segmentación desde la línea de comandos
    float sigma = atof(argv[1]);
    float k = atof(argv[2]);
    int min_size = atoi(argv[3]);

    // Carga la imagen de entrada
    printf("Cargando imagen de entrada desde: %s\n", argv[4]);
    image<rgb> *input = loadPPM(argv[4]);
    printf("Dimensiones de la imagen: %d x %d\n", input->width(), input->height());

    // Inicia el procesamiento y mide el tiempo de ejecución
    printf("Procesando...\n");
    auto start = std::chrono::high_resolution_clock::now();

    int num_ccs;
    // Realiza la segmentación de la imagen usando CUDA
    image<rgb> *seg = segment_image(input, sigma, k, min_size, &num_ccs);

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed = end - start;

    printf("Tiempo de ejecución CUDA: %.6f segundos\n", elapsed.count());

    // Guarda la imagen segmentada en disco
    savePPM(seg, argv[5]);
    printf("Imagen segmentada guardada en: %s\n", argv[5]);
    printf("Número de componentes segmentados: %d\n", num_ccs);

    // Libera la memoria utilizada por las imágenes
    delete input;
    delete seg;

    // Imprime resultados detallados del proceso de segmentación
    printf("\n--- Resultados detallados ---\n");
    printf("Parámetros de segmentación:\n");
    printf("  Sigma: %.2f\n", sigma);
    printf("  K: %.2f\n", k);
    printf("  Min Size: %d\n", min_size);
    printf("Dimensiones de la imagen: %d x %d\n", input->width(), input->height());
    printf("Número de componentes segmentados: %d\n", num_ccs);
    printf("Tiempo total de procesamiento: %.6f segundos\n", elapsed.count());
    printf("-----------------------------\n");

    return 0;
}

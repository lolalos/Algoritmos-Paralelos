
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void testKernel() {
    printf("¡Hola desde la GPU!\n");
}

int main() {
    printf("Ejecutando prueba de CUDA...\n");
    testKernel<<<1, 1>>>();
    hipDeviceSynchronize();
    
    // Verificar dispositivos CUDA
    int deviceCount = 0;
    hipGetDeviceCount(&deviceCount);
    printf("Número de dispositivos CUDA: %d\n", deviceCount);
    
    if (deviceCount > 0) {
        for (int i = 0; i < deviceCount; i++) {
            hipDeviceProp_t deviceProp;
            hipGetDeviceProperties(&deviceProp, i);
            printf("Dispositivo %d: %s\n", i, deviceProp.name);
            printf("  Capacidad de cómputo: %d.%d\n", deviceProp.major, deviceProp.minor);
            printf("  Memoria global: %.2f GB\n", 
                    (float)deviceProp.totalGlobalMem / (1024.0f * 1024.0f * 1024.0f));
        }
        printf("¡CUDA está instalado correctamente!\n");
    } else {
        printf("¡No se encontraron dispositivos CUDA!\n");
    }
    
    return 0;
}